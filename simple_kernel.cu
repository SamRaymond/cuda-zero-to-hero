#include <iostream>
#include <hip/hip_runtime.h>  // Includes CUDA runtime API

// ==============================
// CUDA Kernel Function (Runs on GPU)
// ==============================
/*
    This function runs on the **GPU** (device).
    - __global__ means it's a CUDA kernel and must be launched from the host.
    - It executes in parallel across multiple threads.
    - Each thread has a unique **threadIdx.x**.
*/
__global__ void hello_cuda(int a) {
    int idx = threadIdx.x;  // Get unique thread index within the block

    // Safety check: Limit the number of prints
    if (a > 1000000) a = 1000000;  // Prevent excessive printing

    // Each thread executes this loop independently
    for (int i = 0; i < a; i++) {
        printf("Hello from CUDA thread %d, iteration %d\n", idx, i);
    }
}

// ==============================
// Host Function (Runs on CPU, Launches GPU Kernel)
// ==============================
/*
    This function is callable from Python via SWIG.
    It launches the CUDA kernel on the GPU and synchronizes with the CPU.
*/
extern "C" __host__ void launch_hello_cuda(int a) {
    // Input validation on CPU side
    if (a <= 0) {
        std::cerr << "Error: 'a' must be positive." << std::endl;
        return;
    }

    std::cout << "Launching CUDA kernel from host..." << std::endl;

    // Launch CUDA kernel with 1 block of 10 threads
    hello_cuda<<<1, 10>>>(a);

    // Synchronize CPU and GPU; wait for kernel to complete
    hipError_t err = hipDeviceSynchronize();

    // Error handling: Check for CUDA errors
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}
